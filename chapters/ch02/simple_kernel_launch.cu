/*/
 * 
 *  A simple program to test if cuda is working.
 * 
 *  Compile with:
 *      nvcc simple_kernel_launch.cu
 * 
 *  Run with:
 *     ./a.out
 * source code taken from: https://github.com/R100001/Programming-Massively-Parallel-Processors/blob/master/Chapters/Ch02%20-%20Data%20Parallel%20Computing/labs/simple_kernel_launch/simple_kernel_launch.cu
 * 
/*/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(void) {}

int main() {
    mykernel<<<1,1>>>();
    printf("Hello World!\n");
    return 0;
}