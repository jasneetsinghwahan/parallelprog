#include <stdio.h> 

int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
    printf("  maxBlocksPerMultiprocessor: %d\n", prop.maxBlocksPerMultiProcessor);
    printf("  sharedMemPerBlock : %zu KB\n", prop.sharedMemPerBlock/1024 );
    printf("  sharedMemPerMultiprocessor : %zu KB\n", prop.sharedMemPerMultiprocessor/1024 );
    printf("  unifiedAddressing : %d \n", prop.unifiedAddressing);
    printf("  warpSize : %d\n", prop.warpSize);
    printf("  regsPerBlock : %d\n", prop.regsPerBlock);
    printf("  regsPerMultiprocessor : %d\n", prop.regsPerMultiprocessor);
    printf("  maxThreadsDim : [0] %d [1] %d [2] %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("  maxGridSize : [0] %d [1] %d [2] %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("  compute capability: major.minor: %d.%d\n", prop.major, prop.minor);
    printf("  L2 cache size: %d\n", prop.l2CacheSize);
    printf("  totalConstMem : %zu KB\n", prop.totalConstMem/1024 );
    printf("  totalGlobalMem : %zu GB\n", prop.totalGlobalMem/(1024*1024) );
  }
}